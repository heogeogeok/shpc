#include "hip/hip_runtime.h"
#include "layer.h"

#define div(x, y) (((x) + (y) -1) / (y))

/** SECTION: DEBUGGING **/
#define DEBUG 0
#if DEBUG == 1
double dbg_start_time, dbg_ce_init, dbg_ce_final;
#define DEBUG_PRINT(...) do { \
  printf(__VA_ARGS__); \
} while (0)
#else
#define DEBUG_PRINT(...)
#endif

/** SECTION: GPU manipulation **/
#define NGPU    4
static hipStream_t streams[NGPU];

/** SECTION: Kernels **/
/* Embedding CUDA kernel */
__global__ void kembedding(const int *in, const float *w, float *out, size_t s, size_t H, size_t embedding_dim) {

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x; 

  if (i < s && j < H) { 
    out[i * H + j] = w[in[i] * H + j];
  }
}

/* Permute CUDA kernel */
__global__ void kpermute(const float *in, float *out, size_t s, size_t H) {

  int i = blockIdx.y * blockDim.y + threadIdx.y;  
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < s && j < H) {
    out[j * s + i] = in[i * H + j]; 
  }
}


/* Conv1D CUDA kernel */
__global__ void kconv1d(float *in, float *w, float *b, float *out, 
                              int C, int K, int s, int OC, int os){
    int oc = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y; 

    if (oc < OC && j < os) {
        float val = 0.0f;

        for (int k = 0; k < C; k++) {          
            for (int l = 0; l < K; l++) {      
                val += in[k * s + j + l] * w[oc * C * K + k * K + l];
            }
        }

        val += b[oc];
        out[oc * os + j] = val > 0.0f ? val : 0.0f;
    }
}

/* GetMax CUDA kernel */
__global__ void kgetmax(const float *in, float *out, size_t s, size_t C) {
  
  int i = blockIdx.x * blockDim.x + threadIdx.x;  

  if (i < C) {  
    float max_val = in[i * s]; 
    
    for (size_t j = 1; j < s; j++) {
      max_val = max(max_val, in[i * s + j]);
    }

    out[i] = max_val;
  }
}

/* Concat CUDA kernel */
__global__ void kconcat(const float *in1, const float *in2, const float *in3, const float *in4, 
                              float *out, size_t N1, size_t N2, size_t N3, size_t N4) {
  
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N1) {
    out[idx] = in1[idx];
  } else if (idx < N1 + N2) {
    out[idx] = in2[idx - N1];
  } else if (idx < N1 + N2 + N3) {
    out[idx] = in3[idx - N1 - N2];
  } else if (idx < N1 + N2 + N3 + N4) {
    out[idx] = in4[idx - N1 - N2 - N3];
  }
}

/* Linear CUDA kernel */
__global__ void klinear(float *in, float *w, float *b, float *out, int N, int M, bool relu) {
   
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < M) {
    float val = 0.f;
    for (int j = 0; j < N; j++) {
        val += in[j] * w[i * N + j];
    }

    val += b[i];
    
    if (relu) {
        val = fmaxf(val, 0.0f);
    }
    out[i] = val;
    }
}

/* Embedding
 * @param [in1]  in: [s]
 * @param [in2]   w: [NUM_VOCAB, H]
 * @param [out] out: [s, H]
 * 's' is the sequence length
 * 'H' is the embedding dimension
 */

void Embedding(int *in, Tensor* w, Tensor *out) {
  size_t s = out->shape[0];
  size_t H = out->shape[1];

  int *d_in;
  float *d_w, *d_out;
  size_t in_size = s * sizeof(int);
  size_t w_size = w->shape[0] * w->shape[1] * sizeof(float);
  size_t out_size = s * H * sizeof(float);

  CHECK_CUDA(hipMalloc(&d_in, in_size));
  CHECK_CUDA(hipMalloc(&d_w, w_size));
  CHECK_CUDA(hipMalloc(&d_out, out_size));

  CHECK_CUDA(hipMemcpy(d_in, in, in_size, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_w, w->buf, w_size, hipMemcpyHostToDevice));

  dim3 blockDim(16, 16);
  dim3 gridDim((H + blockDim.x - 1) / blockDim.x, (s + blockDim.y - 1) / blockDim.y);

  kembedding<<<gridDim, blockDim>>>(d_in, d_w, d_out, s, H, w->shape[1]);

  CHECK_CUDA(hipMemcpy(out->buf, d_out, out_size, hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(d_in));
  CHECK_CUDA(hipFree(d_w));
  CHECK_CUDA(hipFree(d_out));
}

/* Permute
 * @param [in]   in: [M, N]
 * @param [out] out: [N, M]
 */
void Permute(Tensor *in, Tensor *out) {
  size_t s = in->shape[0];  
  size_t H = in->shape[1];

  float *d_in, *d_out;
  size_t in_size = s * H * sizeof(float);
  size_t out_size = s * H * sizeof(float);

  CHECK_CUDA(hipMalloc(&d_in, in_size));
  CHECK_CUDA(hipMalloc(&d_out, out_size));

  CHECK_CUDA(hipMemcpy(d_in, in->buf, in_size, hipMemcpyHostToDevice));

  dim3 blockDim(16, 16);
  dim3 gridDim((H + blockDim.x - 1) / blockDim.x, (s + blockDim.y - 1) / blockDim.y);

  kpermute<<<gridDim, blockDim>>>(d_in, d_out, s, H);

  CHECK_CUDA(hipMemcpy(out->buf, d_out, out_size, hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(d_in));
  CHECK_CUDA(hipFree(d_out));
}

/* Conv1D 
 * @param [in1]  in: [C, s]
 * @param [in2]   w: [OC, C, K] 
 * @param [in3]   b: [OC]
 * @param [out] out: [OC, os]
 *    
 *    In this model, K is 3, 5, 7, or 9, 
 *    with stride = 1, pad = 0, dilation = 1.
 *    The formula for the output sequence length:
 *      os = (in - K + 2 * pad) / stride + 1
 *          = (s - K + 2 * 0) / 1 + 1
 *          = s - K + 1
 *
 * 'C' is the input channel size
 * 's' is the input sequence length
 * 'OC' is the output channel size
 * 'os' is the output sequence length
 * 'K' is the kernel (or filter) size
 */
void Conv1D(Tensor *in, Tensor *w, Tensor *b, Tensor *out){

  size_t s = in->shape[1];
  size_t C = in->shape[0];
  size_t OC = w->shape[0];
  size_t K = w->shape[2];

  size_t os = s - K + 1;

  float *d_in, *d_w, *d_b, *d_out;
  size_t size_in = in->num_elem() * sizeof(float);
  size_t size_w = w->num_elem() * sizeof(float);
  size_t size_b = b->num_elem() * sizeof(float);
  size_t size_out = out->num_elem() * sizeof(float);

  for (int i = 0; i < NGPU; ++i) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
  }

  CHECK_CUDA(hipMalloc(&d_in, size_in));
  CHECK_CUDA(hipMalloc(&d_w, size_w));
  CHECK_CUDA(hipMalloc(&d_b, size_b));
  CHECK_CUDA(hipMalloc(&d_out, size_out));

  CHECK_CUDA(hipMemcpy(d_in, in->buf, size_in, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_w, w->buf, size_w, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_b, b->buf, size_b, hipMemcpyHostToDevice));

  dim3 blockDim(16, 16);
  dim3 gridDim((OC + 16) / 16, (os + 16) / 16);

  kconv1d<<<gridDim, blockDim>>>(d_in, d_w, d_b, d_out, C, K, s, OC, os);
  
  CHECK_CUDA(hipMemcpy(out->buf, d_out, size_out, hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(d_in));
  CHECK_CUDA(hipFree(d_w));
  CHECK_CUDA(hipFree(d_b));
  CHECK_CUDA(hipFree(d_out));
}


/* GetMax
 * @param [in]   in: [C, s]
 * @param [out] out: [C]
 *    
 *    This layer is to get the max value along the sequence dim.
 *    The formula for this layer: out = max(in, dim=-1)
 * 
 * 'C' is the channel size
 * 's' is the sequence length
 */
void GetMax(Tensor *in, Tensor *out){
  size_t C = in->shape[0];
  size_t s = in->shape[1];

  float *d_in, *d_out;
  size_t in_size = C * s * sizeof(float);
  size_t out_size = C * sizeof(float);

  hipMalloc(&d_in, in_size);
  hipMalloc(&d_out, out_size);

  hipMemcpy(d_in, in->buf, in_size, hipMemcpyHostToDevice);

  dim3 blockDim(256);  
  dim3 gridDim((C + blockDim.x - 1) / blockDim.x);

  kgetmax<<<gridDim, blockDim>>>(d_in, d_out, s, C);

  hipMemcpy(out->buf, d_out, out_size, hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}

/* Concat
 * @param [in1] in1: [N1]
 * @param [in2] in2: [N2]
 * @param [in3] in3: [N3]
 * @param [in4] in4: [N4]
 * @param [out] out: [N1 + N2 + N3 + N4]
 * 'N1', 'N2', 'N3', and 'N4' are the num of elems in the tensors.
 */
void Concat(Tensor *in1, Tensor *in2, Tensor *in3, Tensor *in4, 
            Tensor *out) {
  
  size_t N1 = in1->shape[0];
  size_t N2 = in2->shape[0];
  size_t N3 = in3->shape[0];
  size_t N4 = in4->shape[0];

  float *d_in1, *d_in2, *d_in3, *d_in4, *d_out;
  size_t out_size = (N1 + N2 + N3 + N4) * sizeof(float);

  CHECK_CUDA(hipMalloc(&d_in1, N1 * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_in2, N2 * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_in3, N3 * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_in4, N4 * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_out, out_size));

  CHECK_CUDA(hipMemcpy(d_in1, in1->buf, N1 * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_in2, in2->buf, N2 * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_in3, in3->buf, N3 * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_in4, in4->buf, N4 * sizeof(float), hipMemcpyHostToDevice));

  dim3 blockDim(256);
  dim3 gridDim((N1 + N2 + N3 + N4 + blockDim.x - 1) / blockDim.x);

  kconcat<<<gridDim, blockDim>>>(d_in1, d_in2, d_in3, d_in4, d_out, N1, N2, N3, N4);

  CHECK_CUDA(hipMemcpy(out->buf, d_out, out_size, hipMemcpyDeviceToHost));
  CHECK_CUDA(hipFree(d_in1));
  CHECK_CUDA(hipFree(d_in2));
  CHECK_CUDA(hipFree(d_in3));
  CHECK_CUDA(hipFree(d_in4));
  CHECK_CUDA(hipFree(d_out));
}

/* Linear 
 * @param [in1]  in: [N]
 * @param [in2]   w: [M, N]
 * @param [in3]   b: [M]
 * @param [out] out: [M]
 * 'N' is the input feature size
 * 'M' is the output feature size
 */
void Linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
    
    float *d_in, *d_w, *d_b, *d_out;
    size_t size_in = in->num_elem() * sizeof(float);
    size_t size_w = w->num_elem() * sizeof(float);
    size_t size_b = b->num_elem() * sizeof(float);
    size_t size_out = out->num_elem() * sizeof(float);

    CHECK_CUDA(hipMalloc(&d_in, size_in));
    CHECK_CUDA(hipMalloc(&d_w, size_w));
    CHECK_CUDA(hipMalloc(&d_b, size_b));
    CHECK_CUDA(hipMalloc(&d_out, size_out));

    CHECK_CUDA(hipMemcpy(d_in, in->buf, size_in, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_w, w->buf, size_w, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, b->buf, size_b, hipMemcpyHostToDevice));

    int M = w->shape[0];
    int blockDim = 256;
    int gridDim = (M + blockDim - 1) / blockDim;
    klinear<<<gridDim, blockDim>>>(d_in, d_w, d_b, d_out, in->shape[0], M, false);

    CHECK_CUDA(hipMemcpy(out->buf, d_out, size_out, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_in));
    CHECK_CUDA(hipFree(d_w));
    CHECK_CUDA(hipFree(d_b));
    CHECK_CUDA(hipFree(d_out));
}

void Linear_ReLU(Tensor *in, Tensor *w, Tensor *b, Tensor *out){
    
    float *d_in, *d_w, *d_b, *d_out;
    size_t size_in = in->num_elem() * sizeof(float);
    size_t size_w = w->num_elem() * sizeof(float);
    size_t size_b = b->num_elem() * sizeof(float);
    size_t size_out = out->num_elem() * sizeof(float);

    CHECK_CUDA(hipMalloc(&d_in, size_in));
    CHECK_CUDA(hipMalloc(&d_w, size_w));
    CHECK_CUDA(hipMalloc(&d_b, size_b));
    CHECK_CUDA(hipMalloc(&d_out, size_out));

    CHECK_CUDA(hipMemcpy(d_in, in->buf, size_in, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_w, w->buf, size_w, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, b->buf, size_b, hipMemcpyHostToDevice));

    int M = w->shape[0];
    int blockDim = 256;
    int gridDim = (M + blockDim - 1) / blockDim;
    klinear<<<gridDim, blockDim>>>(d_in, d_w, d_b, d_out, in->shape[0], M, true);

    CHECK_CUDA(hipMemcpy(out->buf, d_out, size_out, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_in));
    CHECK_CUDA(hipFree(d_w));
    CHECK_CUDA(hipFree(d_b));
    CHECK_CUDA(hipFree(d_out));
}


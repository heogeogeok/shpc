#include "hip/hip_runtime.h"
#include "layer.h"

#define div(x, y) (((x) + (y) -1) / (y))

/** BLOCK SIZE **/
#define LIN_NAIVE_BM 4

#define LIN_REG_BM 16
#define LIN_REG_BK 32

#define C1D_K3_BM 16
#define C1D_K3_BN 8
#define C1D_K3_BK 8

#define C1D_K5_BM 16
#define C1D_K5_BN 8
#define C1D_K5_BK 8

#define C1D_K7_BM 8
#define C1D_K7_BN 32
#define C1D_K7_BK 4

#define C1D_K9_BM 8
#define C1D_K9_BN 32
#define C1D_K9_BK 4

/** KERNELS **/
/* Embedding CUDA kernel */
__global__ void kembedding(const int *in, const float *w, float *out, size_t s, size_t H) {

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x; 
  
  if (i < s && j < H) { 
    out[i * H + j] = w[in[i] * H + j];
  }
}

/* Permute CUDA kernel */
__global__ void kpermute(const float *in, float *out, size_t s, size_t H) {

  int i = blockIdx.y * blockDim.y + threadIdx.y;  
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < s && j < H) {
    out[j * s + i] = in[i * H + j]; 
  }
}

/* Conv1D CUDA kernel */
__global__ void k3conv1d(float *in, float *w, float *b, float *out, 
                              int C, int K, int s, int OC, int os){
  const int BK = C1D_K3_BK;
  const int BN = C1D_K3_BN;
  const int BM = C1D_K3_BM;
  const int KERNEL_SIZE = 3;

  __shared__ float t_in[BK][BN + KERNEL_SIZE - 1 + 4];
  __shared__ float t_w[BM][BK][KERNEL_SIZE + 4];

  float val = 0.0f;

  // output blocks
  int oblock_m_offset = blockIdx.x * BM;
  int oblock_n_offset = blockIdx.y * BN;

  int len_oblock_m = min(BM, OC - oblock_m_offset);
  int len_oblock_n = min(BN, os - oblock_n_offset);

  int othread_m_offset = threadIdx.x / len_oblock_n;
  int othread_n_offset = threadIdx.x % len_oblock_n;

  int othread_valid = othread_m_offset < len_oblock_m;

  for(int bk = 0; bk < C; bk += BK)
  {
    // Load input
    int iblock_k_offset = bk;
    int iblock_n_offset = oblock_n_offset;
    int len_iblock_k = min(BK, C - iblock_k_offset);
    int len_iblock_n = min(BN + KERNEL_SIZE - 1, s - iblock_n_offset);
    int ithread_k_offset = threadIdx.x / len_iblock_n;
    int ithread_n_offset = threadIdx.x % len_iblock_n;

    int ithread_valid = ithread_k_offset < len_iblock_k;

    if (ithread_valid){
      t_in[ithread_k_offset][ithread_n_offset] = in[(iblock_k_offset + ithread_k_offset) * s + iblock_n_offset + ithread_n_offset];
    }

    // Load weight
    int wblock_m_offset = oblock_m_offset;
    int wblock_k_offset = bk;
    int len_wblock_m = min(BM, OC - wblock_m_offset);
    int len_wblock_k = min(BK, C - wblock_k_offset);
    int wthread_m_offset = threadIdx.x / len_wblock_k;
    int wthread_k_offset = threadIdx.x % len_wblock_k;

    int wthread_valid = wthread_m_offset < len_wblock_m;

    if(wthread_valid) {
      for (int i = 0; i < KERNEL_SIZE; i++) {
        t_w[wthread_m_offset][wthread_k_offset][i] = w[(wblock_m_offset + wthread_m_offset) * C * K + (wblock_k_offset + wthread_k_offset) * K + i];
      }
    }

    __syncthreads();

    // Compute
    if (othread_valid) {
      for (int k = 0; k < BK; k++) {
        for (int i = 0; i < KERNEL_SIZE; i++) {
          val += t_w[othread_m_offset][k][i] *  t_in[k][othread_n_offset+ i];
        }  
      }
    }
    
    __syncthreads();
  }

  // Store
  if(othread_valid){
    val += b[oblock_m_offset + othread_m_offset];
    out[(oblock_m_offset + othread_m_offset) * os + oblock_n_offset + othread_n_offset] = val > 0.0f ? val : 0.0f;
  }
}

__global__ void k5conv1d(float *in, float *w, float *b, float *out, 
                              int C, int K, int s, int OC, int os){
  const int BK = C1D_K5_BK;
  const int BN = C1D_K5_BN;
  const int BM = C1D_K5_BM;
  const int KERNEL_SIZE = 5;

  __shared__ float t_in[BK][BN + KERNEL_SIZE - 1 + 4];
  __shared__ float t_w[BM][BK][KERNEL_SIZE + 4];

  float val = 0.0f;

  // output blocks
  int oblock_m_offset = blockIdx.x * BM;
  int oblock_n_offset = blockIdx.y * BN;

  int len_oblock_m = min(BM, OC - oblock_m_offset);
  int len_oblock_n = min(BN, os - oblock_n_offset);

  int othread_m_offset = threadIdx.x / len_oblock_n;
  int othread_n_offset = threadIdx.x % len_oblock_n;

  int othread_valid = othread_m_offset < len_oblock_m;

  for(int bk = 0; bk < C; bk += BK)
  {
    // Load input
    int iblock_k_offset = bk;
    int iblock_n_offset = oblock_n_offset;
    int len_iblock_k = min(BK, C - iblock_k_offset);
    int len_iblock_n = min(BN + KERNEL_SIZE - 1, s - iblock_n_offset);
    int ithread_k_offset = threadIdx.x / len_iblock_n;
    int ithread_n_offset = threadIdx.x % len_iblock_n;

    int ithread_valid = ithread_k_offset < len_iblock_k;

    if (ithread_valid){
      t_in[ithread_k_offset][ithread_n_offset] = in[(iblock_k_offset + ithread_k_offset) * s + iblock_n_offset + ithread_n_offset];
    }

    // Load weight
    int wblock_m_offset = oblock_m_offset;
    int wblock_k_offset = bk;
    int len_wblock_m = min(BM, OC - wblock_m_offset);
    int len_wblock_k = min(BK, C - wblock_k_offset);
    int wthread_m_offset = threadIdx.x / len_wblock_k;
    int wthread_k_offset = threadIdx.x % len_wblock_k;

    int wthread_valid = wthread_m_offset < len_wblock_m;

    if(wthread_valid) {
      for (int i = 0; i < KERNEL_SIZE; i++) {
        t_w[wthread_m_offset][wthread_k_offset][i] = w[(wblock_m_offset + wthread_m_offset) * C * K + (wblock_k_offset + wthread_k_offset) * K + i];
      }
    }

    __syncthreads();

    // Compute
    if (othread_valid) {
      for (int k = 0; k < BK; k++) {
        for (int i = 0; i < KERNEL_SIZE; i++) {
          val += t_w[othread_m_offset][k][i] *  t_in[k][othread_n_offset+ i];
        }  
      }
    }
    
    __syncthreads();
  }

  // Store
  if(othread_valid){
    val += b[oblock_m_offset + othread_m_offset];
    out[(oblock_m_offset + othread_m_offset) * os + oblock_n_offset + othread_n_offset] = val > 0.0f ? val : 0.0f;
  }
}

__global__ void k7conv1d(float *in, float *w, float *b, float *out, 
                              int C, int K, int s, int OC, int os){
  const int BK = C1D_K7_BK;
  const int BN = C1D_K7_BN;
  const int BM = C1D_K7_BM;
  const int KERNEL_SIZE = 7;

  __shared__ float t_in[BK][BN + KERNEL_SIZE - 1 + 4];
  __shared__ float t_w[BM][BK][KERNEL_SIZE + 4];

  float val = 0.0f;

  // Output blocks
  int oblock_m_offset = blockIdx.x * BM;
  int oblock_n_offset = blockIdx.y * BN;

  int len_oblock_m = min(BM, OC - oblock_m_offset);
  int len_oblock_n = min(BN, os - oblock_n_offset);

  int othread_m_offset = threadIdx.x / len_oblock_n;
  int othread_n_offset = threadIdx.x % len_oblock_n;

  int othread_valid = othread_m_offset < len_oblock_m;

  for(int bk = 0; bk < C; bk += BK)
  {
    // Load input
    int iblock_k_offset = bk;
    int iblock_n_offset = oblock_n_offset;
    int len_iblock_k = min(BK, C - iblock_k_offset);
    int len_iblock_n = min(BN + KERNEL_SIZE - 1, s - iblock_n_offset);
    int ithread_k_offset = threadIdx.x / len_iblock_n;
    int ithread_n_offset = threadIdx.x % len_iblock_n;

    int ithread_valid = ithread_k_offset < len_iblock_k;

    if (ithread_valid){
      t_in[ithread_k_offset][ithread_n_offset] = in[(iblock_k_offset + ithread_k_offset) * s + iblock_n_offset + ithread_n_offset];
    }

    // Load weight
    int wblock_m_offset = oblock_m_offset;
    int wblock_k_offset = bk;
    int len_wblock_m = min(BM, OC - wblock_m_offset);
    int len_wblock_k = min(BK, C - wblock_k_offset);
    int wthread_m_offset = threadIdx.x / len_wblock_k;
    int wthread_k_offset = threadIdx.x % len_wblock_k;

    int wthread_valid = wthread_m_offset < len_wblock_m;

    if(wthread_valid) {
      for (int i = 0; i < KERNEL_SIZE; i++) {
        t_w[wthread_m_offset][wthread_k_offset][i] = w[(wblock_m_offset + wthread_m_offset) * C * K + (wblock_k_offset + wthread_k_offset) * K + i];
      }
    }

    __syncthreads();

    // Compute
    if (othread_valid) {
      for (int k = 0; k < BK; k++) {
        for (int i = 0; i < KERNEL_SIZE; i++) {
          val += t_w[othread_m_offset][k][i] *  t_in[k][othread_n_offset+ i];
        }  
      }
    }
    
    __syncthreads();
  }

  // store
  if(othread_valid){
    val += b[oblock_m_offset + othread_m_offset];
    out[(oblock_m_offset + othread_m_offset) * os + oblock_n_offset + othread_n_offset] = val > 0.0f ? val : 0.0f;
  }
}

__global__ void k9conv1d(float *in, float *w, float *b, float *out, 
                              int C, int K, int s, int OC, int os){
  const int BK = C1D_K9_BK;
  const int BN = C1D_K9_BN;
  const int BM = C1D_K9_BM;
  const int KERNEL_SIZE = 9;

  __shared__ float t_in[BK][BN + KERNEL_SIZE - 1 + 4];
  __shared__ float t_w[BM][BK][KERNEL_SIZE + 4];

  float val = 0.0f;

  // output blocks
  int oblock_m_offset = blockIdx.x * BM;
  int oblock_n_offset = blockIdx.y * BN;

  int len_oblock_m = min(BM, OC - oblock_m_offset);
  int len_oblock_n = min(BN, os - oblock_n_offset);

  int othread_m_offset = threadIdx.x / len_oblock_n;
  int othread_n_offset = threadIdx.x % len_oblock_n;

  int othread_valid = othread_m_offset < len_oblock_m;

  for(int bk = 0; bk < C; bk += BK)
  {
    // Load input
    int iblock_k_offset = bk;
    int iblock_n_offset = oblock_n_offset;
    int len_iblock_k = min(BK, C - iblock_k_offset);
    int len_iblock_n = min(BN + KERNEL_SIZE - 1, s - iblock_n_offset);
    int ithread_k_offset = threadIdx.x / len_iblock_n;
    int ithread_n_offset = threadIdx.x % len_iblock_n;

    int ithread_valid = ithread_k_offset < len_iblock_k;

    if (ithread_valid){
      t_in[ithread_k_offset][ithread_n_offset] = in[(iblock_k_offset + ithread_k_offset) * s + iblock_n_offset + ithread_n_offset];
    }

    // Load weight
    int wblock_m_offset = oblock_m_offset;
    int wblock_k_offset = bk;
    int len_wblock_m = min(BM, OC - wblock_m_offset);
    int len_wblock_k = min(BK, C - wblock_k_offset);
    int wthread_m_offset = threadIdx.x / len_wblock_k;
    int wthread_k_offset = threadIdx.x % len_wblock_k;

    int wthread_valid = wthread_m_offset < len_wblock_m;

    if(wthread_valid) {
      for (int i = 0; i < KERNEL_SIZE; i++) {
        t_w[wthread_m_offset][wthread_k_offset][i] = w[(wblock_m_offset + wthread_m_offset) * C * K + (wblock_k_offset + wthread_k_offset) * K + i];
      }
    }

    __syncthreads();

    // Compute
    if (othread_valid) {
      for (int k = 0; k < BK; k++) {
        for (int i = 0; i < KERNEL_SIZE; i++) {
          val += t_w[othread_m_offset][k][i] *  t_in[k][othread_n_offset+ i];
        }  
      }
    }
    
    __syncthreads();
  }

  // store
  if(othread_valid){
    val += b[oblock_m_offset + othread_m_offset];
    out[(oblock_m_offset + othread_m_offset) * os + oblock_n_offset + othread_n_offset] = val > 0.0f ? val : 0.0f;
  }
}


/* GetMax CUDA kernel */
__global__ void kgetmax(const float *in, float *out, size_t s, size_t C) {
  
  int i = blockIdx.x * blockDim.x + threadIdx.x;  

  if (i < C) {  
    float max_val = in[i * s]; 
    
    for (size_t j = 1; j < s; j++) {
      max_val = max(max_val, in[i * s + j]);
    }

    out[i] = max_val;
  }
}

/* Concat CUDA kernel */
__global__ void kconcat(const float *in1, const float *in2, const float *in3, const float *in4, 
                              float *out, size_t N1, size_t N2, size_t N3, size_t N4) {
  
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N1) {
    out[i] = in1[i];
  } else if (i < N1 + N2) {
    out[i] = in2[i - N1];
  } else if (i < N1 + N2 + N3) {
    out[i] = in3[i - N1 - N2];
  } else if (i < N1 + N2 + N3 + N4) {
    out[i] = in4[i - N1 - N2 - N3];
  }
}

/* Linear CUDA kernel */
__global__ void klinear(float *in, float *w, float *b, float *out, int N, int M, bool relu) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < M) {
    float val = 0.f;
    for (int j = 0; j < N; j++) {
        val += in[j] * w[i * N + j];
    }

    val += b[i];
    if (relu) {
        val = fmaxf(val, 0.0f);
    }
    out[i] = val;
  }
}

__global__ void klinear_relu(float *in, float *w, float *b, float *out, int N, int M, bool relu) {
/** CONSTS **/
  const int BM = LIN_REG_BM;
  const int BK = LIN_REG_BK;
  const int LDPT_INPUT = BK / BM;
  const int LDPT_WEIGHT = BK;

  float val = 0.0f;

  int oblock_m = blockIdx.x * BM;

  __shared__ float t_in[BK + 4];
  __shared__ float t_w[BM][BK + 4];

  for (int bk = 0; bk < N; bk += BK) {
    // Load input
    for (int ld_input = 0; ld_input < LDPT_INPUT; ld_input++) {
      t_in[threadIdx.x * LDPT_INPUT + ld_input] = in[bk + threadIdx.x * LDPT_INPUT + ld_input];
    }

    // Load weight
    for (int ld_weight = 0; ld_weight < LDPT_WEIGHT; ld_weight++) {
      t_w[threadIdx.x][ld_weight] = w[N * (oblock_m + threadIdx.x) + bk + ld_weight];
    }

    __syncthreads();

    // Compute
    for (int k = 0; k < BK; k++) {
      val += t_w[threadIdx.x][k] * t_in[k];
    }

    __syncthreads();
  }

  // Store
  val += b[oblock_m + threadIdx.x];
  if (relu && val < 0.0f) val = 0.0f;
  out[oblock_m + threadIdx.x] = val;
}


/* Embedding
 * @param [in1]  in: [s]
 * @param [in2]   w: [NUM_VOCAB, H]
 * @param [out] out: [s, H]
 * 's' is the sequence length
 * 'H' is the embedding dimension
 */

void Embedding(int *in, float* w, float *out, size_t s, size_t H) {
  dim3 blockDim(8, 16);
  dim3 gridDim(div(H, blockDim.x), div(s, blockDim.y));

  kembedding<<<gridDim, blockDim>>>(in, w, out, s, H);
}

/* Permute
 * @param [in]   in: [M, N]
 * @param [out] out: [N, M]
 */
void Permute(float *in, float *out, size_t s, size_t H) {
  dim3 blockDim(32, 8);
  dim3 gridDim(div(H, blockDim.x), div(s, blockDim.y));

  kpermute<<<gridDim, blockDim>>>(in, out, s, H);
}

/* Conv1D 
 * @param [in1]  in: [C, s]
 * @param [in2]   w: [OC, C, K] 
 * @param [in3]   b: [OC]
 * @param [out] out: [OC, os]
 *    
 *    In this model, K is 3, 5, 7, or 9, 
 *    with stride = 1, pad = 0, dilation = 1.
 *    The formula for the output sequence length:
 *      os = (in - K + 2 * pad) / stride + 1
 *          = (s - K + 2 * 0) / 1 + 1
 *          = s - K + 1
 *
 * 'C' is the input channel size
 * 's' is the input sequence length
 * 'OC' is the output channel size
 * 'os' is the output sequence length
 * 'K' is the kernel (or filter) size
 */
void Conv1D_K3(float *in, float *w, float *b, float *out, size_t s, size_t C, size_t OC, size_t K, hipStream_t stream){
  size_t os = s - K + 1;
  dim3 blockDim(C1D_K3_BM * C1D_K3_BN);
  dim3 gridDim(div(OC, C1D_K3_BM), div(os, C1D_K3_BN));

  k3conv1d<<<gridDim, blockDim, 0, stream>>>(in, w, b, out, C, K, s, OC, os);
}

void Conv1D_K5(float *in, float *w, float *b, float *out, size_t s, size_t C, size_t OC, size_t K, hipStream_t stream){
  size_t os = s - K + 1;
  dim3 blockDim(C1D_K5_BM * C1D_K5_BN);
  dim3 gridDim(div(OC, C1D_K5_BM), div(os, C1D_K5_BN));
  k5conv1d<<<gridDim, blockDim, 0, stream>>>(in, w, b, out, C, K, s, OC, os);
}

void Conv1D_K7(float *in, float *w, float *b, float *out, size_t s, size_t C, size_t OC, size_t K, hipStream_t stream){
  size_t os = s - K + 1;
  dim3 blockDim(C1D_K7_BM * C1D_K7_BN);
  dim3 gridDim(div(OC, C1D_K7_BM), div(os, C1D_K7_BN));
  k7conv1d<<<gridDim, blockDim, 0, stream>>>(in, w, b, out, C, K, s, OC, os);
}

void Conv1D_K9(float *in, float *w, float *b, float *out, size_t s, size_t C, size_t OC, size_t K, hipStream_t stream){
  size_t os = s - K + 1;
  dim3 blockDim(C1D_K9_BM * C1D_K9_BN);
  dim3 gridDim(div(OC, C1D_K9_BM), div(os, C1D_K9_BN));
  k9conv1d<<<gridDim, blockDim, 0, stream>>>(in, w, b, out, C, K, s, OC, os);
}

/* GetMax
 * @param [in]   in: [C, s]
 * @param [out] out: [C]
 *    
 *    This layer is to get the max value along the sequence dim.
 *    The formula for this layer: out = max(in, dim=-1)
 * 
 * 'C' is the channel size
 * 's' is the sequence length
 */
void GetMax(float *in, float *out, size_t C, size_t s, hipStream_t stream){
  dim3 blockDim(256);  
  dim3 gridDim(div(C, blockDim.x));
  kgetmax<<<gridDim, blockDim, 0, stream>>>(in, out, s, C);
}

/* Concat
 * @param [in1] in1: [N1]
 * @param [in2] in2: [N2]
 * @param [in3] in3: [N3]
 * @param [in4] in4: [N4]
 * @param [out] out: [N1 + N2 + N3 + N4]
 * 'N1', 'N2', 'N3', and 'N4' are the num of elems in the floats.
 */
void Concat(float *in1, float *in2, float *in3, float *in4, 
            float *out, size_t N1, size_t N2, size_t N3, size_t N4, hipStream_t stream) {
  dim3 blockDim(256);
  dim3 gridDim(div((N1 + N2 + N3 + N4), blockDim.x));
  kconcat<<<gridDim, blockDim, 0, stream>>>(in1, in2, in3, in4, out, N1, N2, N3, N4);
}

/* Linear 
 * @param [in1]  in: [N]
 * @param [in2]   w: [M, N]
 * @param [in3]   b: [M]
 * @param [out] out: [M]
 * 'N' is the input feature size
 * 'M' is the output feature size
 */
void Linear_ReLU(float *in, float *w, float *b, float *out, int N, int M, hipStream_t stream) {
    int blockDim(LIN_REG_BM);
    int gridDim(div(M, blockDim));
    klinear_relu<<<gridDim, blockDim, 0, stream>>>(in, w, b, out, N, M, true);
}

// Final result
void Linear(float *in, float *w, float *b, float *out, int N, int M, hipStream_t stream) {
    int blockDim(LIN_NAIVE_BM);
    int gridDim(div(M, blockDim));
    klinear<<<gridDim, blockDim, 0, stream>>>(in, w, b, out, N, M, false);
}

#include "hip/hip_runtime.h"
#include "layer.h"

#define div(x, y) (((x) + (y) -1) / (y))

/** SECTION: GPU manipulation **/
#define NGPU    4

/** SECTION: Kernels **/
/* ReLU CUDA kernel */
__global__ void krelu(float *inout, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    inout[i] = inout[i] > 0 ? inout[i] : 0;
  }
}

/* Conv1D CUDA kernel */
__global__ void kconv1d(float *in, float *w, float *b, float *out, 
                              int C, int K, int s, int OC, int os){
    int oc = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y; 

    if (oc < OC && j < os) {
        float val = 0.0f;

        for (int k = 0; k < C; k++) {          
            for (int l = 0; l < K; l++) {      
                val += in[k * s + j + l] * w[oc * C * K + k * K + l];
            }
        }

        val += b[oc];
        out[oc * os + j] = val > 0.0f ? val : 0.0f;
    }
}

/* Linear CUDA kernel */
__global__ void klinear(float *in, float *w, float *b, float *out, int N, int M, bool relu) {
   
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < M) {
    float val = 0.f;
    for (int j = 0; j < N; j++) {
        val += in[j] * w[i * N + j];
    }

    val += b[i];
    
    if (relu) {
        val = fmaxf(val, 0.0f);
    }
    

    out[i] = val;
    }
}

/* Embedding
 * @param [in1]  in: [s]
 * @param [in2]   w: [NUM_VOCAB, H]
 * @param [out] out: [s, H]
 * 's' is the sequence length
 * 'H' is the embedding dimension
 */
void Embedding(int *in, Tensor* w, Tensor *out) {
  size_t s = out->shape[0];
  size_t H = out->shape[1];

  #pragma omp parallel for
  for (size_t i = 0; i < s; i++) {
    for (size_t j = 0; j < H; j++) {
      out->buf[i * H + j] = w->buf[in[i] * H + j];
    }
  }
}

/* Permute
 * @param [in]   in: [M, N]
 * @param [out] out: [N, M]
 */
void Permute(Tensor *in, Tensor *out) {
  size_t s = in->shape[0];
  size_t H = in->shape[1];

  // #pragma omp parallel for
  for (size_t i = 0; i < s; i++) {
    for (size_t j = 0; j < H; j++) {
      out->buf[j * s + i] = in->buf[i * H + j];
    }
  }
}

/* Conv1D 
 * @param [in1]  in: [C, s]
 * @param [in2]   w: [OC, C, K] 
 * @param [in3]   b: [OC]
 * @param [out] out: [OC, os]
 *    
 *    In this model, K is 3, 5, 7, or 9, 
 *    with stride = 1, pad = 0, dilation = 1.
 *    The formula for the output sequence length:
 *      os = (in - K + 2 * pad) / stride + 1
 *          = (s - K + 2 * 0) / 1 + 1
 *          = s - K + 1
 *
 * 'C' is the input channel size
 * 's' is the input sequence length
 * 'OC' is the output channel size
 * 'os' is the output sequence length
 * 'K' is the kernel (or filter) size
 */
void Conv1D(Tensor *in, Tensor *w, Tensor *b, Tensor *out){

  size_t s = in->shape[1];
  size_t C = in->shape[0];
  size_t OC = w->shape[0];
  size_t K = w->shape[2];

  size_t os = s - K + 1;

  float *d_in, *d_w, *d_b, *d_out;
  size_t size_in = in->num_elem() * sizeof(float);
  size_t size_w = w->num_elem() * sizeof(float);
  size_t size_b = b->num_elem() * sizeof(float);
  size_t size_out = out->num_elem() * sizeof(float);

  CHECK_CUDA(hipMalloc(&d_in, size_in));
  CHECK_CUDA(hipMalloc(&d_w, size_w));
  CHECK_CUDA(hipMalloc(&d_b, size_b));
  CHECK_CUDA(hipMalloc(&d_out, size_out));

  CHECK_CUDA(hipMemcpy(d_in, in->buf, size_in, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_w, w->buf, size_w, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_b, b->buf, size_b, hipMemcpyHostToDevice));

  dim3 blockDim(16, 16);
  dim3 gridDim((OC + 16) / 16, (os + 16) / 16);

  kconv1d<<<gridDim, blockDim>>>(d_in, d_w, d_b, d_out, C, K, s, OC, os);
  
  CHECK_CUDA(hipMemcpy(out->buf, d_out, size_out, hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(d_in));
  CHECK_CUDA(hipFree(d_w));
  CHECK_CUDA(hipFree(d_b));
  CHECK_CUDA(hipFree(d_out));
}


/* GetMax
 * @param [in]   in: [C, s]
 * @param [out] out: [C]
 *    
 *    This layer is to get the max value along the sequence dim.
 *    The formula for this layer: out = max(in, dim=-1)
 * 
 * 'C' is the channel size
 * 's' is the sequence length
 */
void GetMax(Tensor *in, Tensor *out) {
  size_t C = in->shape[0];
  size_t s = in->shape[1];

  #pragma omp parallel for
  for (size_t i = 0; i < C; i++) {
    out->buf[i] = in->buf[i * s];
    for (size_t j = 1; j < s; j++) {
      out->buf[i] = in->buf[i * s + j] > out->buf[i] ? 
        in->buf[i * s + j] : out->buf[i];
    }
  }
}

/* Concat
 * @param [in1] in1: [N1]
 * @param [in2] in2: [N2]
 * @param [in3] in3: [N3]
 * @param [in4] in4: [N4]
 * @param [out] out: [N1 + N2 + N3 + N4]
 * 'N1', 'N2', 'N3', and 'N4' are the num of elems in the tensors.
 */
void Concat(Tensor *in1, Tensor *in2, Tensor *in3, Tensor *in4, 
            Tensor *out) {
  size_t N1 = in1->shape[0];
  size_t N2 = in2->shape[0];
  size_t N3 = in3->shape[0];
  size_t N4 = in4->shape[0];

  for (size_t i = 0; i < N1; i++) {
    out->buf[i] = in1->buf[i];
  }
  for (size_t i = 0; i < N2; i++) {
    out->buf[N1 + i] = in2->buf[i];
  }
  for (size_t i = 0; i < N3; i++) {
    out->buf[N1 + N2 + i] = in3->buf[i];
  }
  for (size_t i = 0; i < N4; i++) {
    out->buf[N1 + N2 + N3 + i] = in4->buf[i];
  }
}

/* Linear 
 * @param [in1]  in: [N]
 * @param [in2]   w: [M, N]
 * @param [in3]   b: [M]
 * @param [out] out: [M]
 * 'N' is the input feature size
 * 'M' is the output feature size
 */
void Linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
    float *d_in, *d_w, *d_b, *d_out;
    size_t size_in = in->num_elem() * sizeof(float);
    size_t size_w = w->num_elem() * sizeof(float);
    size_t size_b = b->num_elem() * sizeof(float);
    size_t size_out = out->num_elem() * sizeof(float);

    CHECK_CUDA(hipMalloc(&d_in, size_in));
    CHECK_CUDA(hipMalloc(&d_w, size_w));
    CHECK_CUDA(hipMalloc(&d_b, size_b));
    CHECK_CUDA(hipMalloc(&d_out, size_out));

    CHECK_CUDA(hipMemcpy(d_in, in->buf, size_in, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_w, w->buf, size_w, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, b->buf, size_b, hipMemcpyHostToDevice));

    int M = w->shape[0];
    int blockDim = 256;
    int gridDim = (M + blockDim - 1) / blockDim;
    klinear<<<gridDim, blockDim>>>(d_in, d_w, d_b, d_out, in->shape[0], M, false);

    CHECK_CUDA(hipMemcpy(out->buf, d_out, size_out, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_in));
    CHECK_CUDA(hipFree(d_w));
    CHECK_CUDA(hipFree(d_b));
    CHECK_CUDA(hipFree(d_out));
}

void Linear_ReLU(Tensor *in, Tensor *w, Tensor *b, Tensor *out){
    float *d_in, *d_w, *d_b, *d_out;
    size_t size_in = in->num_elem() * sizeof(float);
    size_t size_w = w->num_elem() * sizeof(float);
    size_t size_b = b->num_elem() * sizeof(float);
    size_t size_out = out->num_elem() * sizeof(float);

    CHECK_CUDA(hipMalloc(&d_in, size_in));
    CHECK_CUDA(hipMalloc(&d_w, size_w));
    CHECK_CUDA(hipMalloc(&d_b, size_b));
    CHECK_CUDA(hipMalloc(&d_out, size_out));

    CHECK_CUDA(hipMemcpy(d_in, in->buf, size_in, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_w, w->buf, size_w, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, b->buf, size_b, hipMemcpyHostToDevice));

    int M = w->shape[0];
    int blockDim = 256;
    int gridDim = (M + blockDim - 1) / blockDim;
    klinear<<<gridDim, blockDim>>>(d_in, d_w, d_b, d_out, in->shape[0], M, true);

    CHECK_CUDA(hipMemcpy(out->buf, d_out, size_out, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_in));
    CHECK_CUDA(hipFree(d_w));
    CHECK_CUDA(hipFree(d_b));
    CHECK_CUDA(hipFree(d_out));
}


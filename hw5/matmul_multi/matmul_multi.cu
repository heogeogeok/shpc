#include "hip/hip_runtime.h"
#include "matmul_multi.h"
#include "util.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }

#define MAX_NUM_GPU 4
#define TILESIZE 32

int num_devices = 4;

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                              int K) {
  __shared__ float tA[TILESIZE][TILESIZE];
  __shared__ float tB[TILESIZE][TILESIZE];

  int i = threadIdx.x;
  int j = threadIdx.y;

  int row = i + blockIdx.x * blockDim.x;
  int col = j + blockIdx.y * blockDim.y;

  int ntile = (K + TILESIZE - 1) / TILESIZE;

  float acc = 0.0;
  for (int t = 0; t < ntile; t++)
  {
    int ti = col * K + TILESIZE * t + i;
    int tj = (TILESIZE * t + j) * N + row;
    
    tA[j][i] = A[ti];
    tB[j][i] = B[tj];

    __syncthreads();

    for (int k = 0; k < TILESIZE; k++)
      acc += tA[j][k] * tB[k][i];

    __syncthreads();
  }
  
  C[col * N + row] = acc;
}

// Array of device (GPU) pointers
static float *a_d[MAX_NUM_GPU];
static float *b_d[MAX_NUM_GPU];
static float *c_d[MAX_NUM_GPU];
static int Mbegin[MAX_NUM_GPU], Mend[MAX_NUM_GPU];

void matmul(const float *A, const float *B, float *C, int M, int N, int K) {
  
  #pragma omp parallel for num_threads(num_devices)
  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipMemcpy(a_d[i], A + Mbegin[i] * K,
                          (Mend[i] - Mbegin[i]) * K * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CALL(
          hipMemcpy(b_d[i], B, K * N * sizeof(float), hipMemcpyHostToDevice));

    dim3 threads(TILESIZE, TILESIZE);
    dim3 blocks(N / TILESIZE, ((Mend[i] - Mbegin[i]) / TILESIZE));
    
    CUDA_CALL(hipSetDevice(i));
    matmul_kernel<<<blocks, threads>>>(a_d[i], b_d[i], c_d[i], (Mend[i] - Mbegin[i]), N, K);

    CUDA_CALL(hipMemcpy(C + Mbegin[i] * N, c_d[i],
                          (Mend[i] - Mbegin[i]) * N * sizeof(float),
                          hipMemcpyDeviceToHost));
  }
  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipDeviceSynchronize());
  }
}

void matmul_initialize(int M, int N, int K) {

  CUDA_CALL(hipGetDeviceCount(&num_devices));

  printf("Using %d devices\n", num_devices);
  for (int i = 0; i < num_devices; i++) {
    hipDeviceProp_t prop;
    CUDA_CALL(hipGetDeviceProperties(&prop, i));

    // Try printing more detailed information here
    printf("GPU %d: %s\n", i, prop.name);
  }

  if (num_devices <= 0) {
    printf("No CUDA device found. Aborting\n");
    exit(1);
  }

  // Setup problem size for each GPU
  for (int i = 0; i < num_devices; i++) {
    Mbegin[i] = (M / num_devices) * i;
    Mend[i] = (M / num_devices) * (i + 1);
  }
  Mend[num_devices - 1] = M;

  // Allocate device memory for each GPU
  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipSetDevice(i));
    CUDA_CALL(hipMalloc(&a_d[i], (Mend[i] - Mbegin[i]) * K * sizeof(float)));
    CUDA_CALL(hipMalloc(&b_d[i], K * N * sizeof(float)));
    CUDA_CALL(hipMalloc(&c_d[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
  }
}

void matmul_finalize() {

  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipFree(a_d[i]));
    CUDA_CALL(hipFree(b_d[i]));
    CUDA_CALL(hipFree(c_d[i]));
  }
}
